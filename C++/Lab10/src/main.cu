#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""

#include <iostream>
#include <time.h>
#include <stdlib.h>

#define N 10
#define K 5

__global__ void solve(int* arr, int* result);
void print_arrays(int* A_B);
void initialize_arrays(int* A_B);


int main()
{
    srand(time(NULL));

    int A_B[N + K];
    int result[K];
    
    int* A;
    int* R;

    hipMalloc((void**)&A, (N+K)*sizeof(int));
    hipMalloc((void**)&R, K * sizeof(int));

    initialize_arrays(A_B);
    print_arrays(A_B);

    hipMemcpy(A, A_B, (N + K) * sizeof(int), hipMemcpyHostToDevice);

    solve << < 1, 2 >> > (A, R);

    hipMemcpy(result, R, K * sizeof(int), hipMemcpyDeviceToHost);
    
    for (int i = 0; i < K; i++)
    {
        std::cout << result[i] << " ";
    }

    hipFree(A);
    hipFree(R);

    return 0;
}

__global__ void solve(int* arr, int* result)
{
    int idx = threadIdx.x;

    if (idx == 0)
    {
        for (int i = 0; i < N - 1; i++)
        {
            for (int j = i + 1; j < N; j++)
            {
                if (arr[i] < arr[j])
                {
                    int tmp = arr[j];
                    arr[j] = arr[i];
                    arr[i] = tmp;
                }
            }
        }
    }

    if (idx == 1)
    {
        for (int i = N; i < N + K - 1; i++)
        {
            for (int j = i + 1; j < N + K; j++)
            {
                if (arr[i] < arr[j])
                {
                    int tmp = arr[j];
                    arr[j] = arr[i];
                    arr[i] = tmp;
                }
            }
        }
    }

    __syncthreads();

    for (int i = idx; i < K; i += 2)
    {
        result[i] = arr[i] * arr[N + i];
    }
}

void initialize_arrays(int* A_B)
{
    for (int i = 0; i < N + K; i++)
    {
        A_B[i] = rand() % 10;
    }
}

void print_arrays(int* A_B)
{
    std::cout << "Array A:" << std::endl;
    for (int i = 0; i < N; i++)
    {
        std::cout << A_B[i] << " ";
    }
    std::cout << std::endl;

    std::cout << "Array B:" << std::endl;
    for (int i = 0; i < K; i++)
    {
        std::cout << A_B[N + i] << " ";
    }
    std::cout << std::endl;
}