#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""

#include <iostream>
#include <time.h>
#include <stdlib.h>
#include <stdio.h>

__global__ void calc(int* matrix, int N, int M);

int main()
{
    srand(time(NULL));

    int N = 4;
    int M = 6;


    int* matrix = new int[N * M];

    for (int i = 0; i < N * M; i++)
    {
        matrix[i] = 1 + rand() % 5;
    }

    std::cout << "Matrix:" << std::endl;
    for (int i = 0; i < N; i++)
    {
        for (int j = 0; j < M; j++)
        {
            std::cout << matrix[i * M + j] << " ";
        }
        std::cout << std::endl;
    }
    std::cout << std::endl;

    int* m;
    hipMalloc((void**)&m, (N * M) * sizeof(int));
    hipMemcpy(m, matrix, (N * M) * sizeof(int), hipMemcpyHostToDevice);

    calc << < 1, 2 >> > (m, N, M);

    hipFree(m);
    return 0;
}

__global__ void calc(int* matrix, int N, int M)
{
    int idx = threadIdx.x;

    int* rowSums = new int[N];
    int* colMuls = new int[M];

    if (idx == 0)
    {
        for (int i = 0; i < N; i++)
        {
            int sum = 0;
            for (int j = 0; j < M; j++)
            {
                sum += matrix[i * M + j];
            }
            rowSums[i] = sum;
        }

        printf("Rows sums: ");
        for (int i = 0; i < N; i++)
        {
            printf("%d ", rowSums[i]);
        }
        printf("\n");
    }

    if (idx == 1)
    {
        for (int i = 0; i < M; i++)
        {
            int mul = 1;
            for (int j = 0; j < N; j++)
            {
                mul *= matrix[j * M + i];
            }
            colMuls[i] = mul;
        }

        printf("Columns multiplications: ");
        for (int i = 0; i < M; i++)
        {
            printf("%d ", colMuls[i]);
        }
        printf("\n");
    }
}