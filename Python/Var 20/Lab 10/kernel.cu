#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""

#include <iostream>
#include <time.h>
#include <stdlib.h>
#include <stdio.h>

__global__ void checkMagic(int* matrix, int N, int e);

int main()
{
    srand(time(NULL));

    int N = 0;
    int e = 0;

    std::cout << "Enter N: ";
    std::cin >> N;

    std::cout << "Enter e: ";
    std::cin >> e;

    int* matrix = new int[N * N];

    for (int i = 0; i < N * N; i++)
    {
        matrix[i] = rand() % 5;
    }

    std::cout << "Matrix:" << std::endl;
    for (int i = 0; i < N; i++)
    {
        for (int j = 0; j < N; j++)
        {
            std::cout << matrix[i*N+j] << " ";
        }
        std::cout << std::endl;
    }
    std::cout << std::endl;

    int* m;
    hipMalloc((void**)&m, (N * N) * sizeof(int));
    hipMemcpy(m, matrix, (N * N) * sizeof(int), hipMemcpyHostToDevice);

    checkMagic << < 1, 2 >> > (m, N, e);

    hipFree(m);
    return 0;
}

__global__ void checkMagic(int* matrix, int N, int e)
{
    int idx = threadIdx.x;

    int* horSums = new int[N];
    int* verSums = new int[N];

    if (idx == 0)
    {
        for (int i = 0; i < N; i++)
        {
            int sum = 0;
            for (int j = 0; j < N; j++)
            {
                sum += matrix[i * N + j];
            }
            horSums[i] = sum;
        }

        printf("Horizontal sums: ");
        for (int i = 0; i < N; i++)
        {
            printf("%d ", horSums[i]);
        }
        printf("\n");
    }

    if (idx == 1)
    {
        for (int i = 0; i < N; i++)
        {
            int sum = 0;
            for (int j = 0; j < N; j++)
            {
                sum += matrix[j * N + i];
            }
            verSums[i] = sum;
        }

        printf("Vertical sums: ");
        for (int i = 0; i < N; i++)
        {
            printf("%d ", verSums[i]);
        }
        printf("\n");
    }

    __syncthreads();

    if (idx == 0)
    {
        for (int i = 0; i < N; i++)
        {
            for (int j = 0; j < N; j++)
            {
                if (!(horSums[i] - verSums[j] < e))
                {
                    printf("Not magic");
                    return;
                }
            }
        }
        printf("Magic");
    }
}