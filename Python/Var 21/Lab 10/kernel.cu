#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""

#include <iostream>
#include <stdlib.h>
#include <stdio.h>

__global__ void createMatrix(double* matrix, int stepsX, double x1, double y1, double h);
double checkInput();

int main()
{
    double x1, y1, x2, y2, h;
    std::cout << "x1: ";
    x1 = checkInput();
    std::cout << "x2: ";
    x2 = checkInput();
    std::cout << "y1: ";
    y1 = checkInput();
    std::cout << "y2: ";
    y2 = checkInput();
    std::cout << "Step h: ";
    h = checkInput();

    if (x1 > x2)
    {
        std::swap(x1, x2);
    }

    if (y1 > y2)
    {
        std::swap(y1, y2);
    }

    int stepsX = int((x2 - x1) / h) + 1;
    int stepsY = int((y2 - y1) / h) + 1;

    double* matrix = new double[stepsX * stepsY];

    double x = x1;
    double y = y1;

    double* m;
    hipMalloc((void**)&m, (stepsX * stepsY) * sizeof(double));

    createMatrix << < 1, stepsX * stepsY >> > (m, stepsX, x1, y1, h);

    hipMemcpy(matrix, m, (stepsX * stepsY) * sizeof(double), hipMemcpyDeviceToHost);

    std::cout << "Matrix:" << std::endl;
    for (int i = 0; i < stepsX; i++)
    {
        for (int j = 0; j < stepsX; j++)
        {
            std::cout << matrix[i * stepsX + j] << " ";
        }
        std::cout << std::endl;
    }
    std::cout << std::endl;

    hipFree(m);
    return 0;
}

__global__ void createMatrix(double* m, int stepsX, double x1, double y1, double h)
{
    int idx = threadIdx.x;
    double x = x1 + (idx / stepsX) * h;
    double y = y1 + (idx % stepsX) * h;

    m[idx] = (y - x) * y * cosf(x);
}

double checkInput() 
{
    double a = 0;
    do
    {
        if (!std::cin)
        {
            std::cin.clear();
            while (std::cin.get() != '\n');
            std::cout << "Wrong value, try again\n";
        }
        std::cin >> a;
    } while (!std::cin);

    return a;
}